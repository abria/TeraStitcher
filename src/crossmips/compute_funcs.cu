#include "hip/hip_runtime.h"
//------------------------------------------------------------------------------------------------
// Copyright (c) 2012  Alessandro Bria and Giulio Iannello (University Campus Bio-Medico of Rome).
// Copyright (c) 2017  Massimo Bernaschi (IAC-CNR Roma).  
// All rights reserved.
//------------------------------------------------------------------------------------------------

/*******************************************************************************************************************************************************************************************
*    LICENSE NOTICE
********************************************************************************************************************************************************************************************
*    By downloading/using/running/editing/changing any portion of codes in this package you agree to this license. If you do not agree to this license, do not download/use/run/edit/change
*    this code.
********************************************************************************************************************************************************************************************
*    1. This material is free for non-profit research, but needs a special license for any commercial purpose. Please contact Alessandro Bria at a.bria@unicas.it or Giulio Iannello at
*       g.iannello@unicampus.it for further details.
*    2. You agree to appropriately cite this work in your related studies and publications.
*
*       Bria, A., Iannello, G., "TeraStitcher - A Tool for Fast 3D Automatic Stitching of Teravoxel-sized Microscopy Images", (2012) BMC Bioinformatics, 13 (1), art. no. 316.
*
*    3. This material is provided by  the copyright holders (Alessandro Bria  and  Giulio Iannello),  University Campus Bio-Medico and contributors "as is" and any express or implied war-
*       ranties, including, but  not limited to,  any implied warranties  of merchantability,  non-infringement, or fitness for a particular purpose are  disclaimed. In no event shall the
*       copyright owners, University Campus Bio-Medico, or contributors be liable for any direct, indirect, incidental, special, exemplary, or  consequential  damages  (including, but not
*       limited to, procurement of substitute goods or services; loss of use, data, or profits;reasonable royalties; or business interruption) however caused  and on any theory of liabil-
*       ity, whether in contract, strict liability, or tort  (including negligence or otherwise) arising in any way out of the use of this software,  even if advised of the possibility of
*       such damage.
*    4. Neither the name of University  Campus Bio-Medico of Rome, nor Alessandro Bria and Giulio Iannello, may be used to endorse or  promote products  derived from this software without
*       specific prior written permission.
********************************************************************************************************************************************************************************************/

/******************
*    CHANGELOG    *
*******************
* 2019-04-25. Giulio.     @ADDED using subsampling to compute the first NCC map
* 2019-04-08. Giulio.     @FIXED introduced optimization in loops in function compute_NCC
* 2018-08-29. Giulio.     @CHECKED checked the code implementing the optimization
* 2018.08-27. Giulio.     @FIXED it could happen that auxiliary matrices ps1 and ps2 were not allocated because images are too small and 'seq_cpu_compute_partial_sums' raises an error 
* 2018-08-26. Giulio.     @ADDED code and comments concerning the optimization in computing multiple NCCs on the same images with different overlap
* 2018-06-05. Giulio.     @FIXED bug in allocation of d_missrv which was not reallocated when dimu and/or dimv change
* 2018-05-05. Giulio.     @ADDED conditional code for excluding timing instructions under Windows
* 2018-04-15. Massimo.    @FIXED bugs in CUDA code
* 2018-02-17. Giulio.     @FIXED a bug in 'compute_Neighborhood' when NCCs to be reused are moved internally to the NCC map
* 2017-08-21. Massimo.    @ADDED   CUDA implementation of NCC
* 2017-04-01. Giulio.     @CHANGED the algorithm that computes the peak width
* 2015-04-06. Giulio.     @CHANGED corrected compute_NCC_alignment to deal with the case widthX = 1 which likely to be an anomaly
* 2015-03-20. Giulio.     @CHANGED newu and newv have been moved as parameters in compute_Neighborhood
* 2014-10-31. Giulio.     @CHANGED computations in compute_NCC are performed in double precision (and not in single precision) to avoit roundoff errors
*/

/*
 * compute_funcs.cpp
 *
 *  Created on: September 2010
 *      Author: iannello
 *
 *  Last revision: May, 31 2011
 */

# include <math.h>
# include <stdio.h>
# include <stdlib.h>
# include <assert.h>

#ifdef USE_OPENMP
        #include <omp.h>
#endif

#ifndef TIMING_H
#define TIMING_H

#ifdef _WIN32

#define TIMER_DEF     

#define TIMER_START   

#define TIMER_STOP    

#define TIMER_ELAPSED 0    // this must be an expression

#else

#include <sys/time.h>

#define TIMER_DEF     struct timeval temp_1, temp_2

#define TIMER_START   gettimeofday(&temp_1, (struct timezone*)0)

#define TIMER_STOP    gettimeofday(&temp_2, (struct timezone*)0)

#define TIMER_ELAPSED ((temp_2.tv_sec-temp_1.tv_sec)*1.e6+(temp_2.tv_usec-temp_1 .tv_usec))

#endif

#endif // TIMING_H

//extern double simpletimer;
double simpletimer;

/********* CONTROL SYMBOLS AND VARIABLES FOR CUDA CODE ****************************************************/

/* the USECUDA symbol controls conditional compilation of CUDA code 
 * it is defined by CMake when the WITH_CUDA option is checked
 * when WITH_CUDA option is unchecked, this file is included by compute_funcs.cpp to force C++ compilation
 */

// NCC is computed on NVIDIA card if the environment variable USECUDA_X_NCC exists 
#define CUDAENVVAR "USECUDA_X_NCC"

/* this variable control actual execution on NVIDIA card It can assume the following values: 
 * -1: non initialized
 * 0: do not use NVIDIA card
 * 1: use NVIDIA card
 */
static int usecuda=-1; 
/*********************************************************************************************************/


# include "compute_funcs.h"

# define LOG2(V)   (log((double)V)/log(2.0))


/**************************** INTERNAL AUXILIARY FUNCTION *******************************/

/* search x in the sorted list (a,n) and returns if x has been found; if x is not found,
 * returns in pos the index at which x should be inserted (i.e. a[pos] is the first element
 * of the list that is greater than x)
 * specialized for use by enhance: returns the index of the linear transformation to be
 * applied to value x
 */
static
void binary_search ( iom::real_t *a, int n, iom::real_t x, bool &found, int &pos ) {
        int f = 0;
        int l = n-1;
        int m;

        found = false;
        while ( f <= l ) {
                m = (f+l)/2;
                if ( x == a[m] ) {
                        found = true;
                        pos = m+1; // if found, the index to be returned is the next one
                        return;
                }
                else if ( x < a[m] )
                        l = m - 1;
                else   // V > a[m]
                        f = m + 1;
        }
        pos = l+1;
        return;
}

/* given general parameters, the NCC map, its second dimension dimj, the index of NCC maximum
 * and the extension of the map along the vertical (wRangeThr1) and horizontal (wRangeThr2) directions
 * returns a measure of the vertical (width1) and horizontal (width2) half widths of the NCC peak
 * (i.e. the largest distance of the pixel equal to a given fraction of the maximum and the
 * maximum itself)
 */
static
void compute_NCC_width ( NCC_parms_t *NCC_params, iom::real_t *NCC, int dimj, int ind, int wRangeThr1, int wRangeThr2, bool failed, int &width1, int &width2 ) {
        bool found;
        iom::real_t prec_val;
        int dist;

        iom::real_t thr = NCC_params->widthThr * NCC[ind];

        if(failed)
        {
                width1 = width2 = NCC_params->INF_W;
        }
        else
        {
                // evaluates first maximum width parallel to second dimension (horizontal)
                if ( wRangeThr2 < NCC_params->minDim_NCCmap ) {
                        // the map is too narrow in horizontal direction
                        width2 = NCC_params->INF_W;
                }
                else {
                        // try to find a very clear peak
                        found = false;
                        width2 = 1;
                        while ( width2<=wRangeThr2 && !found )
                                if ( NCC[ind-width2] <= thr )
                                        found = true;
                                else
                                        width2++;
                        found = false;
                        while ( width2<=wRangeThr2 && !found )
                                if ( NCC[ind+width2] <= thr )
                                        found = true;
                                else
                                        width2++;
                        if ( !found ) { // try to find if there is a peak anyway
                                // skip NCC_params->minPoints points
                                prec_val = NCC[ind-NCC_params->minPoints];
                                dist = NCC_params->minPoints + 1;
                                while ( dist<=wRangeThr2 && !found )
                                        if ( NCC[ind-dist] >= prec_val )
                                                found = true;
                                        else {
                                                prec_val = NCC[ind-dist];
                                                dist++;
                                        }
                                if ( dist < (2*NCC_params->minPoints) ) // not enough points
                                        width2 = NCC_params->INF_W;
                                else
                                        // project the profile to compute the equivalent width at thr
                                        width2 = (int)floor((dist-1) * (NCC[ind] - thr) / (NCC[ind] - prec_val));

                                found = false;
                                prec_val = NCC[ind+NCC_params->minPoints];
                                dist = NCC_params->minPoints + 1;
                                while ( dist<=wRangeThr2 && !found ) {
                                        if ( NCC[ind+dist] >= prec_val ) // NCC increases
                                                found = true;
                                        else {
                                                prec_val = NCC[ind+dist];
                                                dist++;
                                        }
                                }
                                if ( dist < (2*NCC_params->minPoints) ) // not enough points
                                        width2 = NCC_params->INF_W;
                                else
                                        // project the profile to compute the equivalent width at thr and compare with the largest width that is not 'infinite'
                                        width2 = MIN(MAX(width2,(int)floor((dist-1) * (NCC[ind] - thr) / (NCC[ind] - prec_val))),NCC_params->INF_W-1);
                        }
                }

                // evaluates maximum width parallel to first dimension (vertical)
                if ( wRangeThr1 < NCC_params->minDim_NCCmap ) {
                        // the map is too narrow in vertical direction
                        width1 = NCC_params->INF_W;
                }
                else {
                        // try to find a very clear peak
                        found = false;
                        width1 = 1;
                        while ( width1<=wRangeThr1 && !found )
                                if ( NCC[ind-width1*dimj] <= thr )
                                        found = true;
                                else
                                        width1++;
                        found = false;
                        while ( width1<=wRangeThr1 && !found )
                                if ( NCC[ind+width1*dimj] <= thr )
                                        found = true;
                                else
                                        width1++;
                        if ( !found ) { // try to find if there is a peak anyway
                                prec_val = NCC[ind-NCC_params->minPoints*dimj];
                                dist = NCC_params->minPoints + 1;
                                while ( dist<=wRangeThr2 && !found )
                                        if ( NCC[ind-dist*dimj] >= prec_val )
                                                found = true;
                                        else {
                                                prec_val = NCC[ind-dist*dimj];
                                                dist++;
                                        }
                                if ( dist < (2*NCC_params->minPoints) ) // not enough points
                                        width1 = NCC_params->INF_W;
                                else
                                        width1 = (int)floor((dist-1) * (NCC[ind] - thr) / (NCC[ind] - prec_val));

                                found = false;
                                prec_val = NCC[ind+NCC_params->minPoints*dimj];
                                dist = NCC_params->minPoints + 1;
                                while ( dist<=wRangeThr2 && !found )
                                        if ( NCC[ind+dist*dimj] >= prec_val )
                                                found = true;
                                        else {
                                                prec_val = NCC[ind+dist*dimj];
                                                dist++;
                                        }
                                if ( dist < (2*NCC_params->minPoints) ) // not enough points
                                        width1 = NCC_params->INF_W;
                                else
                                        // project the profile to compute the equivalent width at thr and compare with the largest width that is not 'infinite'
                                        width1 = MIN(MAX(width1,(int)floor((dist-1) * (NCC[ind] - thr) / (NCC[ind] - prec_val))),NCC_params->INF_W-1);
                        }
                }
        }
}

/*************************** COMPUTE FINAL ALIGNMENT *****************************************/

/* given the general parameters, the index i of a dimension (0 for vertical, 1 for horizontal
 * and 2 for depth), and two alignments d1 and d2 with the corresponding NCC maxima (peak_val1
 * and peak_val2) and half widths (width1 and width2), returns in result the aligment for that
 * with a measure of its reliability and potential error
 *
 * 2015-04-06. Giulio. The algorithm has been vcorrected as follows:
 * when widthX is 1 the alignment is unreliable since it is very likely that it is due to
 * a spike or a too little NCC map (e.g. because the stack is very thin); for this reason
 * parametes width1 and width2 are first checked and changes if equal to 1
 */
static
void compute_NCC_alignment ( NCC_parms_t *NCC_params, NCC_descr_t *result, int i,
                                                        int d1, iom::real_t peak_val1, int width1, int d2, iom::real_t peak_val2, int width2 ) {

        // check width1 and widthw
        if ( width1 == 1 ) // alignment 1 is unreliable
                width1 = NCC_params->INF_W;
        if ( width2 == 1 ) // alignment 1 is unreliable
                width2 = NCC_params->INF_W;

        // check how many values contribute to final alignment
        if ( peak_val1 >= NCC_params->maxThr && width1 < NCC_params->INF_W ) // first value may be considered
                if ( peak_val2 >= NCC_params->maxThr && width2 < NCC_params->INF_W ) // second value may be considered too
                        if ( abs(d1 - d2) < MIN(width1,width2) ) { // both values must be considered
                                result->coord[i] = (int) floor((peak_val1 * d1 + peak_val2 * d2) / (peak_val1 + peak_val2) + 0.5); // weighted mean of alignments
                                result->NCC_maxs[i] = (peak_val1 * peak_val1 + peak_val2 * peak_val2) / (peak_val1 + peak_val2); // weighted mean of reliabilities
                                result->NCC_widths[i] = MAX(width1,width2); // maximum width
                        }
                        else { // only one value should be considered: take into account both peak value and peak width
                                if ( peak_val1/width1 > peak_val2/width2 ) { // first value should be considered
                                        result->coord[i] = d1;
                                        result->NCC_maxs[i] = peak_val1;
                                        result->NCC_widths[i] = width1;
                                }
                                else { // second value should be considered
                                        result->coord[i] = d2;
                                        result->NCC_maxs[i] = peak_val2;
                                        result->NCC_widths[i] = width2;
                                }
                        }
                else { // only first value should be considered
                        result->coord[i] = d1;
                        result->NCC_maxs[i] = peak_val1;
                        result->NCC_widths[i] = width1;
                }
        else
                if ( peak_val2 >= NCC_params->maxThr && width2 < NCC_params->INF_W ) { // only second value should be considered
                        result->coord[i] = d2;
                        result->NCC_maxs[i] = peak_val2;
                        result->NCC_widths[i] = width2;
                }
                else { // none value is reliable
                        result->coord[i] = NCC_params->INV_COORD;  // invalid coordinate
                        result->NCC_maxs[i] = NCC_params->UNR_NCC; // unreliable NCC
                        result->NCC_widths[i] = NCC_params->INF_W; // peak of infinite width
                }
}
/****************************************************************************************/



/*********** THREADS PAREMETERS AND CODE ************/

# if defined(_PAR_VERSION) && 0  // WARNING: the code enabled by symbol _PAR_VERSION is obsolete and not fully checked

# include <windows.h>

// parallel configuration ------------------------------
int n_procs = 0;
int par_degree = 0;

void init_configuration ( ) {
        fprintf(stdout,"--- PARALLEL VERSION RUNNING ---\n");
        n_procs = atoi(getenv("NUMBER_OF_PROCESSORS"));
        par_degree = n_procs;
}

// compute_3_MIPs --------------------------------------

typedef struct{
        // input parametres
        iom::real_t *A;
        iom::real_t *B;
        int dimi_v;
        int dimj_v;
        int dimk_v;
        int MIP_stridek;
        int stridei;
        int stridek;
        // input/ouput parameters
        // output parametres
        iom::real_t *MIP_xy1;
        iom::real_t *MIP_xz1;
        iom::real_t *MIP_yz1;
        iom::real_t *MIP_xy2;
        iom::real_t *MIP_xz2;
        iom::real_t *MIP_yz2;
} compute_3_MIPs_params_t;

DWORD WINAPI compute_3_MIPs_worker ( LPVOID lpParam ) {
        iom::real_t *A       = ((compute_3_MIPs_params_t *) lpParam)->A;
        iom::real_t *B       = ((compute_3_MIPs_params_t *) lpParam)->B;
        int dimi_v      = ((compute_3_MIPs_params_t *) lpParam)->dimi_v;
        int dimj_v      = ((compute_3_MIPs_params_t *) lpParam)->dimj_v;
        int dimk_v      = ((compute_3_MIPs_params_t *) lpParam)->dimk_v;
        int stridei     = ((compute_3_MIPs_params_t *) lpParam)->stridei;
        int stridek     = ((compute_3_MIPs_params_t *) lpParam)->stridek;
        int MIP_stridek = ((compute_3_MIPs_params_t *) lpParam)->MIP_stridek;
        iom::real_t *MIP_xy1 = ((compute_3_MIPs_params_t *) lpParam)->MIP_xy1;
        iom::real_t *MIP_xz1 = ((compute_3_MIPs_params_t *) lpParam)->MIP_xz1;
        iom::real_t *MIP_yz1 = ((compute_3_MIPs_params_t *) lpParam)->MIP_yz1;
        iom::real_t *MIP_xy2 = ((compute_3_MIPs_params_t *) lpParam)->MIP_xy2;
        iom::real_t *MIP_xz2 = ((compute_3_MIPs_params_t *) lpParam)->MIP_xz2;
        iom::real_t *MIP_yz2 = ((compute_3_MIPs_params_t *) lpParam)->MIP_yz2;

        iom::real_t *vol1, *vol2;
        int i, j, k;

        // calcola MIP su xy, xz, yz scandendo una sola volta i due volumi
        for ( k=0, vol1=A, vol2=B; k<dimk_v; k++, vol1+=stridek, vol2+=stridek )
                for ( i=0; i<dimi_v; i++, vol1+=stridei, vol2+=stridei )
                        for ( j=0; j<dimj_v; j++, vol1++, vol2++ ) {
                                MIP_xy1[i*dimj_v+j] = MAX(MIP_xy1[i*dimj_v+j],*vol1);
                                MIP_xz1[i*MIP_stridek+k] = MAX(MIP_xz1[i*MIP_stridek+k],*vol1); // MIP stride along k dimension is the original MIP k dimension
                                MIP_yz1[j*MIP_stridek+k] = MAX(MIP_yz1[j*MIP_stridek+k],*vol1); // MIP stride along k dimension is the original MIP k dimension
                                MIP_xy2[i*dimj_v+j] = MAX(MIP_xy2[i*dimj_v+j],*vol2);
                                MIP_xz2[i*MIP_stridek+k] = MAX(MIP_xz2[i*MIP_stridek+k],*vol2); // MIP stride along k dimension is the original MIP k dimension
                                MIP_yz2[j*MIP_stridek+k] = MAX(MIP_yz2[j*MIP_stridek+k],*vol2); // MIP stride along k dimension is the original MIP k dimension
                        }

        return 0;
}

// compute_NCC_map --------------------------------------

typedef struct{
        // input parametres
        iom::real_t *MIP_1;
        iom::real_t *MIP_2;
        int dimu;
        int dimv;
        int delayu;
        int delayv;
        int u_start;
        int v_start;
        int u_end;
        int v_end;
        // input/ouput parameters
        // output parametres
        iom::real_t *NCC_map;
} compute_NCC_map_params_t;

DWORD WINAPI compute_NCC_map_worker ( LPVOID lpParam ) {
        iom::real_t *MIP_1    = ((compute_NCC_map_params_t *) lpParam)->MIP_1;
        iom::real_t *MIP_2    = ((compute_NCC_map_params_t *) lpParam)->MIP_2;
        int dimu         = ((compute_NCC_map_params_t *) lpParam)->dimu;
        int dimv         = ((compute_NCC_map_params_t *) lpParam)->dimv;
        int delayu       = ((compute_NCC_map_params_t *) lpParam)->delayu;
        int delayv       = ((compute_NCC_map_params_t *) lpParam)->delayv;
        int u_start      = ((compute_NCC_map_params_t *) lpParam)->u_start;
        int v_start      = ((compute_NCC_map_params_t *) lpParam)->v_start;
        int u_end        = ((compute_NCC_map_params_t *) lpParam)->u_end;
        int v_end        = ((compute_NCC_map_params_t *) lpParam)->v_end;
        iom::real_t *NCC_map  = ((compute_NCC_map_params_t *) lpParam)->NCC_map;

        iom::real_t *im1, *im2;
        int u, v;

        // nel seguito u=0 rappresenta il massimo scostamento negativo del secondo MIP rispetto al primo
        // con riferimento alla prima coordinata; v=0 ha il medesimo significato con riferimento alla seconda
        // coordinata

        for ( u=u_start; u<=u_end; u++ )
                for ( v=v_start; v<=v_end; v++ ) {
                        im1 = MIP_1 + START_IND(u*dimv) + START_IND(v);
                        im2 = MIP_2 + START_IND(-u*dimv) + START_IND(-v);
                        NCC_map[(u+delayu)*(2*delayv+1)+(v+delayv)] = compute_NCC(im1,im2,dimu-abs(u),dimv-abs(v),abs(v));

                }

        return 0;
}

# endif // defined(_PAR_VERSION) && 0


/************ OPERATIONS IMPLEMENTATION *************/

void seq_cpu_compute_partial_sums(iom::real_t *image1, iom::real_t *image2, int height, int width, 
								  iom::real_t *ps1, iom::real_t *ps2){
	
	// 2018. Giulio. @ADDED if auxiliary matrices have not been allocated do nothing
	if ( !ps1 || !ps2 )
		return;
		
	// assumes that ps1 and ps2 are allocated and have size floor(height/TILE_SIDE) x floor(width/TILE_SIDE)
    int nh = height - (height % TILE_SIDE);
    int nw = width  - (width % TILE_SIDE);

    // int ph = nh / TILE_SIDE; // Giulio: not used?
    int pw = nw / TILE_SIDE;

    for(int i = 0; i < nh; i += TILE_SIDE){
        for(int j = 0; j < nw; j+= TILE_SIDE){
            ps1[(i / TILE_SIDE)*pw + (j/TILE_SIDE)] = 0;
            ps2[(i / TILE_SIDE)*pw + (j/TILE_SIDE)] = 0;
            for(int l = 0; l < TILE_SIDE; l++){
                for (int k = 0; k < TILE_SIDE; k++){                    
                    ps1[(i / TILE_SIDE)*pw + (j/TILE_SIDE)] += image1[(i + l) * width + (j + k)];
                    ps2[(i / TILE_SIDE)*pw + (j/TILE_SIDE)] += image2[(i + l) * width + (j + k)];
                }
            }
        }
    }
}

void compute_3_MIPs ( iom::real_t *A, iom::real_t *B,
                                          iom::real_t *MIP_xy1, iom::real_t *MIP_xz1, iom::real_t *MIP_yz1,
                                          iom::real_t *MIP_xy2, iom::real_t *MIP_xz2, iom::real_t *MIP_yz2,
                                          int dimi_v, int dimj_v, int dimk_v, int stridei, int stridek ) {
# ifndef _PAR_VERSION 

        iom::real_t *vol1, *vol2;
        int i, j, k;

        // calcola MIP su xy, xz, yz scandendo una sola volta i due volumi
        for ( k=0, vol1=A, vol2=B; k<dimk_v; k++, vol1+=stridek, vol2+=stridek )
                for ( i=0; i<dimi_v; i++, vol1+=stridei, vol2+=stridei )
                        for ( j=0; j<dimj_v; j++, vol1++, vol2++ ) {
                                MIP_xy1[i*dimj_v+j] = MAX(MIP_xy1[i*dimj_v+j],*vol1);
                                MIP_xz1[i*dimk_v+k] = MAX(MIP_xz1[i*dimk_v+k],*vol1);
                                MIP_yz1[j*dimk_v+k] = MAX(MIP_yz1[j*dimk_v+k],*vol1);
                                MIP_xy2[i*dimj_v+j] = MAX(MIP_xy2[i*dimj_v+j],*vol2);
                                MIP_xz2[i*dimk_v+k] = MAX(MIP_xz2[i*dimk_v+k],*vol2);
                                MIP_yz2[j*dimk_v+k] = MAX(MIP_yz2[j*dimk_v+k],*vol2);
                        }

# else // ndef _PAR_VERSION     WARNING: the code enabled by symbol _PAR_VERSION is obsolete and not fully checked

        HANDLE *workerHandles = new HANDLE[par_degree];
        compute_3_MIPs_params_t *compute_3_MIPs_params = new compute_3_MIPs_params_t[par_degree];
        int t, i, j;

        /*
         *  work decomposition is performed by partitioning the volum along the k (i.e. z) direction
         *  each thread compute a portion of MIPS in xz and yz planes and a partial MIP in xy plane
         *  partial MIPS are then merged
         */

        // partition dimk_v
        int n1 = dimk_v / par_degree;
        int n2 = dimk_v % par_degree;
        for ( t=0; t<n2; t++ )
                compute_3_MIPs_params[t].dimk_v = n1 + 1;
        for ( ; t<par_degree; t++ )
                compute_3_MIPs_params[t].dimk_v = n1;

        // allocate and initialize memory for partial MIP computation
        iom::real_t **MIP_xy1_lst = new iom::real_t *[par_degree];
        iom::real_t **MIP_xy2_lst = new iom::real_t *[par_degree];
        // first partial MIPs are stored in MIP_xy1 and MIP_xy2
        MIP_xy1_lst[0] = MIP_xy1;
        MIP_xy2_lst[0] = MIP_xy2;
        for ( t=1; t<par_degree; t++ ) {
                MIP_xy1_lst[t] = new iom::real_t[dimi_v*dimj_v];
                MIP_xy2_lst[t] = new iom::real_t[dimi_v*dimj_v];
        }

        int slice_dim = dimi_v*(dimj_v+stridei)+stridek;
        /*
         * number of pixels of one slice of volumes A and B
         *
         * case NORTH_SOUTH:
         *    dimi_v  = dimi - ni
         *    dimj_v  = dimj
         *    stridei = 0
         *    stridek = ni*dimj    ====>
         *      slice_dim = (dimi - ni) * dimj + ni * dimj = dimi * dimj
         *
         * case WEST_EAST:
         *    dimi_v  = dimi
         *    dimj_v  = dimj - nj
         *    stridei = nj
         *    stridek = 0          ====>
         *      slice_dim = dimi * ( dimj - nj + nj) + 0 = dimi * dimj
         */
        int depth = 0;
        for ( t=0; t<par_degree; t++ ) {
                compute_3_MIPs_params[t].A = A + depth*slice_dim;
                compute_3_MIPs_params[t].B = B + depth*slice_dim;
                compute_3_MIPs_params[t].dimi_v = dimi_v;
                compute_3_MIPs_params[t].dimj_v = dimj_v;
                compute_3_MIPs_params[t].stridei = stridei;
                compute_3_MIPs_params[t].stridek = stridek;
                compute_3_MIPs_params[t].MIP_stridek = dimk_v;
                compute_3_MIPs_params[t].MIP_xy1 = MIP_xy1_lst[t];
                compute_3_MIPs_params[t].MIP_xz1 = MIP_xz1 + depth;
                compute_3_MIPs_params[t].MIP_yz1 = MIP_yz1 + depth;
                compute_3_MIPs_params[t].MIP_xy2 = MIP_xy2_lst[t];
                compute_3_MIPs_params[t].MIP_xz2 = MIP_xz2 + depth;
                compute_3_MIPs_params[t].MIP_yz2 = MIP_yz2 + depth;

                workerHandles[t] = CreateThread( NULL, 0, compute_3_MIPs_worker, (compute_3_MIPs_params+t), 0, NULL);

                depth += compute_3_MIPs_params[t].dimk_v;
        }

        WaitForMultipleObjects(par_degree,workerHandles,TRUE,INFINITE);

        for ( t=0; t<par_degree; t++ )
                CloseHandle(workerHandles[t]);

        // compute global MIP_xy
        for ( t=1; t<par_degree; t++ )
                for ( i=0; i<dimi_v; i++ )
                        for ( j=0; j<dimj_v; j++ ) {
                                MIP_xy1[i*dimj_v + j] = MAX(MIP_xy1[i*dimj_v + j],MIP_xy1_lst[t][i*dimj_v + j]);
                                MIP_xy2[i*dimj_v + j] = MAX(MIP_xy2[i*dimj_v + j],MIP_xy2_lst[t][i*dimj_v + j]);
                        }

        // deallocation
        for ( t=1; t<par_degree; t++ ) {
                delete MIP_xy1_lst[t];
                delete MIP_xy2_lst[t];
        }
        delete MIP_xy1_lst;
        delete MIP_xy2_lst;

        delete compute_3_MIPs_params;
        delete workerHandles;
        
# endif // ndef _PAR_VERSION
}

#if defined(USECUDA)
static iom::real_t *d_im1=NULL, *d_im2=NULL, *d_rv=NULL, *d_missrv=NULL;
static iom::real_t *dev_ps1 = NULL; // partial sums of image 1 (sums optimization)
static iom::real_t *dev_ps2 = NULL; // partial sums of image 2 (sums optimization)
static int sizeps = -1;             // size auxiliary matrices (sums optimization)
static int *d_missu=NULL, *d_missv=NULL;
static iom::real_t *s_im1=NULL, *s_im2=NULL;
static unsigned int sizeimg=0, sizeout=0;
static int sizemiss = -1;
static int sizerv = -1; // 2018-06-05. Giulio. @ADDED variable d_missrv_size to correctly initialize d_missrv

#include "warp_reduce.h"

#define MY_CUDA_CHECK( call) {                                    \
    hipError_t err = call;                                                    \
    if( hipSuccess != err) {                                                \
        fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n",        \
                __FILE__, __LINE__, hipGetErrorString( err) );              \
        exit(EXIT_FAILURE);                                                  \
    } }

#define MY_CHECK_ERROR(errorMessage) {                                    \
    hipError_t err = hipGetLastError();                                    \
    if( hipSuccess != err) {                                                \
        fprintf(stderr, "Cuda error: %s in file '%s' in line %i : %s.\n",    \
                errorMessage, __FILE__, __LINE__, hipGetErrorString( err) );\
        exit(EXIT_FAILURE);                                                  \
    }                                                                        \
    }

#define NSTREAMS 2
#define NTHREADS 1024
#define MAXBLOCKS 1
#define REAL float
//#define USE_LDG
#ifdef USE_LDG
#define LDG(x) (__ldg(&(x)))
#else
#define LDG(x) (x)
#endif

#define CALCSUM(v) (v)=warpReduceSumD((v)); \
        if(lane==0) shared[wid]=(v); \
        __syncthreads(); \
        (v) = (threadIdx.x<blockDim.x/warpSize) ? shared[lane] : 0; \
        if(wid==0) (v)=warpReduceSumD((v)); \
        if(tid==0) shared[0]=(v); \
        __syncthreads(); \
        (v)=shared[0]; \
        __syncthreads();

#define CALCAVE(v) CALCSUM(v) \
        (v) /= (dimi*dimj);

__global__ void compute_partial_sums(iom::real_t *image1, iom::real_t *image2, int height, int width, iom::real_t *ps1, iom::real_t *ps2, int tileSide){

        int tid_x = blockDim.x * blockIdx.x + threadIdx.x;
        int tid_y = blockDim.y * blockIdx.y + threadIdx.y;
    
        int th = blockDim.x * gridDim.x;
        int tw = blockDim.y * gridDim.y; 
        
        __shared__ double partials1[NTHREADS];
        __shared__ double partials2[NTHREADS];
        
        int nh = height - (height % tileSide); // assumption: height, width >= tileSide
        int nw = width  - (width % tileSide);
        int pw = nw / tileSide;
        int threadSpan = tileSide / blockDim.x;
    
        
        
    
        for (int x = tid_x * threadSpan; x < nh; x += th*threadSpan){
            for(int y = tid_y * threadSpan; y < nw; y += tw*threadSpan){
                // Each loop of this while computes the sum of a tile
                double v1 = 0;
                double v2 = 0;
                for (int i = x;  i < x + threadSpan; i += 1){
                    for(int j = y; j < y + threadSpan; j += 1){
                        v1 += image1[i * width + j];
                        v2 += image2[i * width + j];
                    }
                }
                            
                // local coordinates with respect to block
                int tid = threadIdx.x * blockDim.y + threadIdx.y;
                partials1[tid] = v1;
                partials2[tid] = v2;
                __syncthreads();
    
                // reduce sum insde a warp
                for(int i = NTHREADS / 2; i > 0; i /= 2){
                    if(tid < i){
                        partials1[tid] += partials1[tid + i];
                        partials2[tid] += partials2[tid + i];           
                    }
                    __syncthreads();
                }
            
                if(tid == 0){
                    ps1[(x / tileSide)*pw + (y/tileSide)] = (float) partials1[0];
                    ps2[(x / tileSide)*pw + (y/tileSide)] = (float) partials2[0];
                }
            } 
        }
    }
    
            
            
__global__ void
__launch_bounds__(1024, 1)
gpu_NCC_map ( iom::real_t *MIP_1, iom::real_t *MIP_2, iom::real_t *rv, int dimu, int dimv, int delayu, int delayv,
        iom::real_t *ps1, iom::real_t *ps2, int tileSide ) {

        static __shared__ double shared[32];
     
        double f_mean, t_mean, f_prime, t_prime, numerator, factor1, factor2;

        int u=-delayu+blockIdx.x;
        int v=-delayv+blockIdx.y;
        int dimi=dimu-abs(u);
        int dimj=dimv-abs(v);
        int stride=abs(v);

        int a_u = START_IND(u);
        int a_v = START_IND(v);
        int b_u = START_IND(-u);
        int b_v = START_IND(-v);
        
        iom::real_t *pxl1 = MIP_1 + dimv * a_u + a_v;
        iom::real_t *pxl2 = MIP_2 + b_u*dimv + b_v;

        /*
        iom::real_t *pxl1 = MIP_1 + START_IND(u*dimv) + START_IND(v);
        iom::real_t *pxl2 = MIP_2 + START_IND(-u*dimv) + START_IND(-v);
        */
        const unsigned int tid = threadIdx.x;

        int lane=tid%warpSize;
        int wid=tid/warpSize;

        unsigned int ij;

        f_mean = t_mean = 0;
       
        // --- Cristian's addition: START
        
        // see comments in function 'NNC_compute' for further documentation
        // WARNING: in this implementation it is assumed that ps1 and ps2 have been allocated and initialized
        //          actually ps1 and ps2 are initialized with global CUDA variables dev_ps1 and dev_ps2
        
        int pw = (dimv / tileSide);
        int blockSide = 32;
        int i, j;
        int tid_x = (threadIdx.x / blockSide);
        int tid_y = (threadIdx.x % blockSide);
        
         // start indices of the overlapping region of MIP_1 and MIP_2 assigned to this thread
        int start_a_i = tid_x + a_u;
        int start_a_j = tid_y + a_v;
        int start_b_i = tid_x + b_u;
        int start_b_j = tid_y + b_v;

        // Indexes defining the tiled region inside the overlapping region of MIP_1
        int start_tiled_block_a_u = a_u - (a_u % tileSide);
        int start_tiled_block_a_v = a_v - (a_v % tileSide);
        start_tiled_block_a_u = start_tiled_block_a_u == a_u ? start_tiled_block_a_u : start_tiled_block_a_u + tileSide;
        start_tiled_block_a_v = start_tiled_block_a_v == a_v ? start_tiled_block_a_v : start_tiled_block_a_v + tileSide;
        int end_tiled_block_a_u = a_u + dimi - ((a_u + dimi) % tileSide);
        int end_tiled_block_a_v = a_v + dimj - ((a_v + dimj) % tileSide);

        // Indexes defining the tiled region inside the overlapping region of MIP_2
        int start_tiled_block_b_u = b_u - (b_u % tileSide);
        int start_tiled_block_b_v = b_v - (b_v % tileSide);
        start_tiled_block_b_u = start_tiled_block_b_u == b_u ? start_tiled_block_b_u : start_tiled_block_b_u + tileSide;
        start_tiled_block_b_v = start_tiled_block_b_v == b_v ? start_tiled_block_b_v : start_tiled_block_b_v + tileSide;
        int end_tiled_block_b_u = b_u + dimi - ((b_u + dimi) % tileSide);
        int end_tiled_block_b_v = b_v + dimj - ((b_v + dimj) % tileSide);

        // Since continuous threads doesnt work on contiguous pixels in this part, we advance with a larger offset
        int newBlockSide = blockSide * tileSide;
        // First, sum all the values in the tiled region. We will move the threads only on special pixels of 
        // the overlapping region, that is, the ones with indexes that are multiples of tileSide and are inside the
        // tiled region of the overlapping region. (In a local coordinates of a tile, they are at position (0, 0))
        
        // first for image MIP_1
        for(i = start_tiled_block_a_u + tileSide*tid_x; i < end_tiled_block_a_u; i+= newBlockSide){
                int row = (i/tileSide)*pw;
                for(j = start_tiled_block_a_v + tileSide*tid_y; j < end_tiled_block_a_v; j+= newBlockSide){
                f_mean += ps1[ row + (j/tileSide)];
                }
        }
        // then for image MIP_2
        for(i = start_tiled_block_b_u + tileSide*tid_x; i < end_tiled_block_b_u; i+= newBlockSide){
                int row = (i/tileSide)*pw;
                for(j = start_tiled_block_b_v + tileSide*tid_y; j < end_tiled_block_b_v; j+= newBlockSide){
                t_mean += ps2[ row + (j/tileSide)];
                }
        }

        // now sum the pixels at the border, that is, all the ones that are not inside the tiled region
        for(i = start_a_i; i < dimi + a_u; i+= blockSide){
                j = start_a_j;
                while(j < dimj + a_v){
                    if(j < start_tiled_block_a_v || j >= end_tiled_block_a_v || i < start_tiled_block_a_u || i >= end_tiled_block_a_u){
                        f_mean += MIP_1[i*dimv + j];
                        j+= blockSide;
                    }else{
                        int pixels_to_jump = end_tiled_block_a_v - j;
                        j += ((pixels_to_jump / blockSide) + ((pixels_to_jump % blockSide) ? 1 : 0))*blockSide;
                    }
                }
            }
        
            for(i = start_b_i; i < dimi + b_u; i+= blockSide){
                j = start_b_j;
                while(j < dimj + b_v) {
                    if(j < start_tiled_block_b_v || j >= end_tiled_block_b_v || i < start_tiled_block_b_u || i >= end_tiled_block_b_u){
                        t_mean += MIP_2[i*dimv + j];
                        j+= blockSide;
                    }else{
                        int pixels_to_jump = end_tiled_block_b_v - j;
                        j += ((pixels_to_jump / blockSide) + ((pixels_to_jump % blockSide) ? 1 : 0))*blockSide;
                    }
                }
            }
        // --- Cristian's addition: END
        /*
        for ( ij=tid; ij<(dimi*dimj); ij+=blockDim.x) {
          f_mean+=LDG(pxl1[(ij%dimj)+(stride+dimj)*(ij/dimj)]);
          t_mean+=LDG(pxl2[(ij%dimj)+(stride+dimj)*(ij/dimj)]);
        }*/
        CALCAVE(f_mean);
        CALCAVE(t_mean);
        
      //if(tid==0)  printf("FMEAN: %f, TMEAN: %f\n",f_mean, t_mean);
        // applies the optimization at the beginning of section 5 of Lewis article (t_prime has zero mean)
        numerator = 0;
        factor1 = 0;
        factor2 = 0;
        for ( ij=tid; ij<(dimi*dimj); ij+=blockDim.x) {
          f_prime = LDG(pxl1[(ij%dimj)+(stride+dimj)*(ij/dimj)]) - f_mean;
          t_prime = LDG(pxl2[(ij%dimj)+(stride+dimj)*(ij/dimj)]) - t_mean;
          numerator += LDG(pxl1[(ij%dimj)+(stride+dimj)*(ij/dimj)]) * t_prime;
          factor1 += f_prime * f_prime;
          factor2 += t_prime * t_prime;
        }
        CALCSUM(numerator);
        CALCSUM(factor1);
        CALCSUM(factor2);

        if(tid==0) {
//        printf("NUMERATOR: %f, FACTOR1: %f, FACTOR2: %f\n",numerator, factor1, factor2);
          rv[(u+delayu)*(2*delayv+1)+(v+delayv)]=((float) (numerator / sqrt(factor1*factor2))); // the result is converted to single precision
        }

}

__global__ void
__launch_bounds__(1024, 1)
gpu_NCC_miss ( iom::real_t *MIP_1, iom::real_t *MIP_2, iom::real_t *missrv,
              int dimu, int dimv, int du, int dv,
              int newu, int newv, int *missu, int *missv) {

        static __shared__ double shared[32];

        double f_mean, t_mean, f_prime, t_prime, numerator, factor1, factor2;

        int u = missu[blockIdx.x] - newu + du;
        int v = missv[blockIdx.x] - newv + dv;
        int dimi=dimu-abs(u);
        int dimj=dimv-abs(v);

        if(START_IND(dimi) == 0 || START_IND(dimj) == 0)
                return;

        int stride=abs(v);
        iom::real_t *pxl1 = MIP_1 + START_IND(u*dimv) + START_IND(v);
        iom::real_t *pxl2 = MIP_2 + START_IND(-u*dimv) + START_IND(-v);

        const unsigned int tid = threadIdx.x;

        int lane=tid%warpSize;
        int wid=tid/warpSize;

        unsigned int ij;
        f_mean = t_mean = 0;
        for ( ij=tid; ij<(dimi*dimj); ij+=blockDim.x) {
          f_mean+=pxl1[(ij%dimj)+(stride+dimj)*(ij/dimj)];
          t_mean+=pxl2[(ij%dimj)+(stride+dimj)*(ij/dimj)];
        }
        CALCAVE(f_mean);
        CALCAVE(t_mean);
//      if(tid==0)  printf("FMEAN: %f, TMEAN: %f\n",f_mean, t_mean);
        // applies the optimization at the beginning of section 5 of Lewis article (t_prime has zero mean)
        numerator = 0;
        factor1 = 0;
        factor2 = 0;
        for ( ij=tid; ij<(dimi*dimj); ij+=blockDim.x) {
          f_prime = pxl1[(ij%dimj)+(stride+dimj)*(ij/dimj)] - f_mean;
          t_prime = pxl2[(ij%dimj)+(stride+dimj)*(ij/dimj)] - t_mean;
          numerator += pxl1[(ij%dimj)+(stride+dimj)*(ij/dimj)] * t_prime;
          factor1 += f_prime * f_prime;
          factor2 += t_prime * t_prime;
        }
        CALCSUM(numerator);
        CALCSUM(factor1);
        CALCSUM(factor2);

        if(tid==0) {
//        printf("NUMERATOR: %f, FACTOR1: %f, FACTOR2: %f\n",numerator, factor1, factor2);
          missrv[missu[blockIdx.x]*(2*newv+1)+missv[blockIdx.x]]=((float) (numerator / sqrt(factor1*factor2))); // the result is converted to single precision
        }

}

#endif

void compute_NCC_map ( iom::real_t *NCC_map, iom::real_t *MIP_1, iom::real_t *MIP_2,
                                               int dimu, int dimv, int delayu, int delayv ) {
	//# ifndef _PAR_VERSION

# if !defined(_PAR_VERSION)

        if(usecuda<0) { // check if NVIDIA card should be used
           usecuda=getenv(CUDAENVVAR)?1:0;
        }

#if defined(USECUDA) // 2019-04-25. Giulio. Check 'usecuda' variable only if CUDA code is enabled

        if(usecuda>0) { // use the NVIDIA card
        
			unsigned int nthreads=NTHREADS;
			dim3 dimGrid((2*delayu)+1,(2*delayv+1));
			// computes means
			hipEvent_t start, stop;
			hipEventCreate(&start);
			hipEventCreate(&stop);
			hipEventRecord( start, 0 );
		
			// ------ cristian's addition START
			dim3 dimThread(32, 32);
			dim3 dimGridPS(8, 8);
			int tileSide = 32;
		
			int ph = dimu / tileSide;
			int pw = dimv / tileSide;
			int newSizePS =  pw * ph;  // total size of the matrix containing the partial sums (i.e. sums of each tile)
			if(sizeps < newSizePS){
					if(sizeps > 0){
							hipFree(dev_ps1);
							hipFree(dev_ps2);
					}
					sizeps = newSizePS;
					MY_CUDA_CHECK ( hipMalloc((void **)&dev_ps1, sizeof(iom::real_t) * sizeps));
					MY_CUDA_CHECK ( hipMalloc((void **)&dev_ps2, sizeof(iom::real_t) * sizeps));
			}               
			// ------ cristian's addition END

			if(sizeimg<(dimu*dimv)) {
			  if(sizeimg>0) {
				MY_CUDA_CHECK(hipFree(d_im1));
				MY_CUDA_CHECK(hipFree(d_im2));
			  }
			  sizeimg=dimu*dimv;
			  MY_CUDA_CHECK ( hipMalloc (( void **) &d_im1, sizeimg*sizeof ( iom::real_t ) ));
			  MY_CUDA_CHECK ( hipMalloc (( void **) &d_im2, sizeimg*sizeof ( iom::real_t ) ));
			}
			if(sizeout<((2*delayu+1)*(2*delayv+1))) {
			  if(sizeout>0) {
				MY_CUDA_CHECK(hipFree(d_rv));
			  }
			  sizeout=(2*delayu+1)*(2*delayv+1);
			  MY_CUDA_CHECK ( hipMalloc (( void **) &d_rv, sizeout*sizeof ( iom::real_t ) ));

			}
			if(s_im1!=MIP_1) {
					MY_CUDA_CHECK( hipMemcpy(d_im1, MIP_1,
								   sizeof(iom::real_t)*(dimu*dimv), hipMemcpyHostToDevice) );
					s_im1=MIP_1;
			}
			if(s_im2!=MIP_2) {
					MY_CUDA_CHECK( hipMemcpy(d_im2, MIP_2,
								   sizeof(iom::real_t)*(dimu*dimv), hipMemcpyHostToDevice) );
					s_im2=MIP_2;
			}
			MY_CUDA_CHECK( hipMemcpy(d_rv, NCC_map,
					 sizeof(iom::real_t)*(2*delayu+1)*(2*delayv+1), hipMemcpyHostToDevice) );

			// --- Cristian's addition START 
			compute_partial_sums<<<dimGridPS, dimThread>>>(d_im1, d_im2, dimu, dimv, dev_ps1, dev_ps2, tileSide);
			gpu_NCC_map<<<dimGrid,nthreads>>>(d_im1, d_im2, d_rv, dimu, dimv, delayu, delayv, dev_ps1, dev_ps2, tileSide);
			// ---- Cristian's addition END

			MY_CUDA_CHECK( hipMemcpy(NCC_map, d_rv,
					 sizeof(iom::real_t)*(2*delayu+1)*(2*delayv+1), hipMemcpyDeviceToHost) );
			hipEventRecord( stop, 0 );
			hipEventSynchronize( stop );
			float elapsedTime;
			MY_CUDA_CHECK( hipEventElapsedTime( &elapsedTime, start, stop ) );
			simpletimer += elapsedTime*1000;        
        } 
        else 
        
#endif //defined(USECUDA)
        { // 2019-04-25. Giulio. If CUDA code is not enabled this code should be executed in any case
        	TIMER_DEF;   
        	TIMER_START;
        	int ph = dimu / TILE_SIDE;
        	int pw = dimv / TILE_SIDE;
			int psmatrix_size =  ph * pw; // total size of the matrix containing the partial sums (i.e. sums of each tile)
			iom::real_t *ps1 = (iom::real_t *) 0;
			iom::real_t *ps2 = (iom::real_t *) 0;
        	if(psmatrix_size > 0){
				ps1 = (iom::real_t *) malloc(sizeof(iom::real_t) * psmatrix_size);
				ps2 = (iom::real_t *) malloc(sizeof(iom::real_t) * psmatrix_size);
				if(!ps1 || !ps2){
					// TODO: handle malloc failures according to policy

				}
			}
	
			seq_cpu_compute_partial_sums(MIP_1, MIP_2, dimu, dimv, ps1, ps2);

#ifdef USE_OPENMP // WARNING: OpenMP not sufficiently tested and not maintaine
        	
        	// assume processors are a power of 2
			int nProcessors = omp_get_max_threads();
    	    int a = 2;
        	while(a*a < nProcessors) a*=2;
        	int b = nProcessors / a;
        	omp_set_num_threads(a*b);
        	#pragma omp parallel
        	{
        		int u, v;
        		int start_u = omp_get_thread_num() / b;
        		int start_v = omp_get_thread_num() % b;

        		for ( u=-delayu + start_u; u<=delayu; u += a)
                	for ( v=-delayv + start_v; v<=delayv; v += b) {
                        NCC_map[(u+delayu)*(2*delayv+1)+(v+delayv)] = compute_NCC(MIP_1,MIP_2,dimu,dimv, u, v, ps1, ps2);
                	}
        	}

#else // USE_OPENMP
        	
	        int u, v;
        	for ( u=-delayu; u<=delayu; u++ ) 
            	for ( v=-delayv; v<=delayv; v++ ) {
// 2019-04-25. Giulio. @CHANGED use subsampling for finding the maximum 
//         	for ( u=-delayu; u<=delayu; u+=2 ) 
//             	for ( v=-delayv; v<=delayv; v+=2 ) { 
                	NCC_map[(u+delayu)*(2*delayv+1)+(v+delayv)] = compute_NCC(MIP_1,MIP_2,dimu,dimv, u, v, ps1, ps2);
                }
                
#endif // USE_OPENMP
        	
        	if(ps1) 
        		delete ps1;
        	if(ps2) 
        		delete ps2;
        		
        	TIMER_STOP; 
        	simpletimer+=TIMER_ELAPSED;
        } // endif defined(USECUDA)
        
#if defined(DUMPNCC)

        for ( int u=-delayu; u<=delayu; u++ ) {
                for ( int v=-delayv; v<=delayv; v++ ) {
                   fprintf(stderr,"NCC_map[%d]=%f\n", (u+delayu)*(2*delayv+1)+(v+delayv),
                                        NCC_map[(u+delayu)*(2*delayv+1)+(v+delayv)]);
                }
        }
        
#endif // defined(DUMPNCC)

# else //!defined(_PAR_VERSION) WARNING: the code enabled by symbol _PAR_VERSION is obsolete and not fully checked

        int npu, npv, nu1, nu2, nv1, nv2;
        int pu, u, pv, v, t, du, dv;

        // assume the processors are a power of 2
        int pow_2 = (int) floor(LOG2(par_degree));
        int par_degree_2 = (int) pow(2.0,pow_2);

        HANDLE *workerHandles = new HANDLE[par_degree_2];
        compute_NCC_map_params_t *compute_NCC_map_params = new compute_NCC_map_params_t[par_degree_2];

        /*
         *  work decomposition is done by partitioning the NCC maps
         *  maps are halved repeatedly until floor(log2(#procs)) blocks are generated
         *  each thread is given the indices of one block and computes the NCCs corresponding to that block
         */

        npv = pow_2 / 2;
        npu = npv + (pow_2 % 2);
        npv = (int) pow(2.0,npv); // partitions along v
        npu = (int) pow(2.0,npu); // partitions along u

        nu1 = (2*delayu+1) / npu; // minimum number of rows per partition along u
        nu2 = (2*delayu+1) % npu; // number of partitions along u that have one more row
        nv1 = (2*delayv+1) / npv; // minimum number of rows per partition along v
        nv2 = (2*delayu+1) % npv; // number of partitions along v that have one more column

        for ( pu=0, u=-delayu, t=0; pu<npu; pu++, u+=du ) {
                du = (pu<nu2) ? (nu1+1) : nu1;
                for ( pv=0, v=-delayv ; pv<npv; pv++, v+=dv, t++ ) {
                        dv = (pv<nv2) ? (nv1+1) : nv1;
                        compute_NCC_map_params[t].u_start = u;
                        compute_NCC_map_params[t].v_start = v;
                        compute_NCC_map_params[t].u_end = u + (du-1);
                        compute_NCC_map_params[t].v_end = v + (dv-1);
                }
        }

        for ( t=0; t<par_degree_2; t++ ) {
                compute_NCC_map_params[t].MIP_1 = MIP_1;
                compute_NCC_map_params[t].MIP_2 = MIP_2;
                compute_NCC_map_params[t].dimu = dimu;
                compute_NCC_map_params[t].dimv = dimv;
                compute_NCC_map_params[t].delayu = delayu;
                compute_NCC_map_params[t].delayv = delayv;
                compute_NCC_map_params[t].NCC_map = NCC_map;

                workerHandles[t] = CreateThread( NULL, 0, compute_NCC_map_worker, (compute_NCC_map_params+t), 0, NULL);
        }

        WaitForMultipleObjects(par_degree_2,workerHandles,TRUE,INFINITE);

        for ( t=0; t<par_degree_2; t++ )
                CloseHandle(workerHandles[t]);

        // deallocation
        delete compute_NCC_map_params;
        delete workerHandles;

# endif // !defined(_PAR_VERSION)

}


iom::real_t compute_NCC ( iom::real_t *MIP_1, iom::real_t *MIP_2, int dimu, int dimv, int u, int v, iom::real_t *ps1, iom::real_t *ps2) {
	// parallelization of compute_NCC_map makes parallelization of this operation pointless
    // 2014-10-31 Giulio. @CHANGED       iom::real_t f_mean, t_mean, f_prime, t_prime, numerator, factor1, factor2;
    double f_mean, t_mean, f_prime, t_prime, numerator, factor1, factor2;
    iom::real_t *pxl1, *pxl2;
    int i, j;

	int dimi = dimu - abs(u); // number of rows of the overlapping region
	int dimj = dimv - abs(v); // number of columns of the overlapping region
	int stride = abs(v);

	// these variables are needed by the code that is executed when optimization is not applied
	iom::real_t* im1 = MIP_1 + START_IND(u*dimv) + START_IND(v);
	iom::real_t* im2 = MIP_2 + START_IND(-u*dimv) + START_IND(-v);

    // computes means
    f_mean = t_mean = 0;
        
	// if auxiliary matrices have not been allocated do nothing
	if ( ps1 && ps2 && TILE_SIDE <= dimu && TILE_SIDE <= dimv ){ 
		// 2018.08-27. Giulio. FIXED ps1 and ps2 must be allocated AND both dimu and dimv must be larger than TILE_SIDE
		// ps1 and ps2 have been allocated and at least the sums of some tiles have already been computed and stored in ps1 and ps2

		// START_IND(i) = (((i)>0) ? (i) : 0)
		int pw = (dimv / TILE_SIDE); // width of matrix of precomputed sums
		int a_u = START_IND(u);  // u index over MIP_1 of overlapping region
		int a_v = START_IND(v);  // v index over MIP_1 of overlapping region
		int b_u = START_IND(-u); // u index over MIP_2 of overlapping region
		int b_v = START_IND(-v); // v index over MIP_2 of overlapping region


		// Indexes defining the tiled region inside the overlapping region of MIP_1
		int start_tiled_block_a_u = a_u - (a_u % TILE_SIDE); // u index of first tile intersecting the overlapping region
		int start_tiled_block_a_v = a_v - (a_v % TILE_SIDE); // v index of first tile intersecting the overlapping region
		// if indices are outside the overlapping region they are advanced by TILE_SIDE
		start_tiled_block_a_u = start_tiled_block_a_u == a_u ? start_tiled_block_a_u : start_tiled_block_a_u + TILE_SIDE;
		start_tiled_block_a_v = start_tiled_block_a_v == a_v ? start_tiled_block_a_v : start_tiled_block_a_v + TILE_SIDE;
		int end_tiled_block_a_u = a_u + dimi - ((a_u + dimi) % TILE_SIDE); // u index following the last one of a tile internal to the overlapping region
		int end_tiled_block_a_v = a_v + dimj - ((a_v + dimj) % TILE_SIDE); // v index following the last one of a tile internal to the overlapping region

		// Indexes defining the tiled region inside the overlapping region of MIP_2
		// the same comments of MIP_1 hold
		int start_tiled_block_b_u = b_u - (b_u % TILE_SIDE);
		int start_tiled_block_b_v = b_v - (b_v % TILE_SIDE);
		start_tiled_block_b_u = start_tiled_block_b_u == b_u ? start_tiled_block_b_u : start_tiled_block_b_u + TILE_SIDE;
		start_tiled_block_b_v = start_tiled_block_b_v == b_v ? start_tiled_block_b_v : start_tiled_block_b_v + TILE_SIDE;
		int end_tiled_block_b_u = b_u + dimi - ((b_u + dimi) % TILE_SIDE);
		int end_tiled_block_b_v = b_v + dimj - ((b_v + dimj) % TILE_SIDE);

		// First, sum all the values in the tiled region. We will move the threads only on special pixels of 
		// the overlapping region, that is, the ones with indexes that are multiples of TILE_SIDE and are inside the
		// tiled region of the overlapping region. (In a local coordinates of a tile, they are at position (0, 0))
		
		// first for image MIP_1
		// i, j move to the first vertex of tiles internal to the overlapping region
		for(i = start_tiled_block_a_u; i  < end_tiled_block_a_u; i+= TILE_SIDE){
			int row = (i/TILE_SIDE)*pw;
			for(j = start_tiled_block_a_v; j < end_tiled_block_a_v; j+= TILE_SIDE){
				f_mean += ps1[ row + (j/TILE_SIDE)];
			}
		}
		// then for image MIP_2
		// i, j move to the first vertex of tiles internal to the overlapping region
		for(i = start_tiled_block_b_u; i < end_tiled_block_b_u; i+= TILE_SIDE){
			int row = (i/TILE_SIDE)*pw;
			for(j = start_tiled_block_b_v; j < end_tiled_block_b_v; j+= TILE_SIDE){
				t_mean += ps2[ row + (j/TILE_SIDE)];
			}
		}

		// first for MIP_!
		// now sum the pixels at the border, that is, all the ones that are not inside the tiled region
		// each pixel of the overlapping region is processed at most once (it is not processed if it is in the tiled region
		for(i = a_u; i < dimi + a_u; i+= 1){
			j = a_v;
			while(j < dimj + a_v){
				if(j < start_tiled_block_a_v || j >= end_tiled_block_a_v || i < start_tiled_block_a_u || i >= end_tiled_block_a_u){
					f_mean += MIP_1[i*dimv + j];
					j+= 1;
				}else{ // index j is just entered in the tiled region: skip the tiled region
					int pixels_to_jump = end_tiled_block_a_v - j;
					j += pixels_to_jump;
				}
			}
		}
		
		// then for MIP_2
		// the same comments of MIP_1 hold
		for(i = b_u; i < dimi + b_u; i+= 1){
			j = b_v;
			while(j < dimj + b_v) {
				if(j < start_tiled_block_b_v || j >= end_tiled_block_b_v || i < start_tiled_block_b_u || i >= end_tiled_block_b_u){
					t_mean += MIP_2[i*dimv + j];
					j+= 1;
				}else{
					int pixels_to_jump = end_tiled_block_b_v - j;
					j += pixels_to_jump;
				}
			}
		}
	}else{
		// code without optimization
		for ( i=0, pxl1=im1, pxl2=im2; i<dimi; i++, pxl1+=stride, pxl2+=stride ) {
			for ( j=0; j<dimj; j++, pxl1++, pxl2++ ) {
				f_mean += *pxl1;
				t_mean += *pxl2;
			}
		}
	}

    f_mean /= (dimi*dimj);
    t_mean /= (dimi*dimj);

	// applies the optimization at the beginning of section 5 of Lewis article (t_prime has zero mean)
	numerator = 0;
	factor1 = 0;
	factor2 = 0;
	// 2019-04-08. Giulio. using two moving pointers optimizes the double loop
	for ( i=0, pxl1=im1, pxl2=im2; i<dimi; i++, pxl1+=stride, pxl2+=stride )
		for ( j=0; j<dimj; j++, pxl1++, pxl2++ ) {
			f_prime = *pxl1 - f_mean;
			t_prime = *pxl2 - t_mean;
			numerator += *pxl1 * t_prime;
			factor1 += f_prime * f_prime;
			factor2 += t_prime * t_prime;
		}

    return ((float) (numerator / sqrt(factor1*factor2))); // the result is converted to single precision

}

int compute_MAX_ind ( iom::real_t *vect, int len ) {
// actual len values are too small to deserve parallelization
        int i;
        iom::real_t val_max = vect[0];
        int ind_max = 0;
        for ( i=0; i<len; i++ )
                if ( vect[i] > val_max ) {
                        val_max = vect[i];
                        ind_max = i;
                }
        return ind_max;
}


/* Returns an NCC map of of size (2*newu+1) x (2*newv+1) centered around the NCC maximum or returns filed = true
 * if this map cannot be found (it is: newu = newv = NCC_params->wRangeThr)
 *
 * Parameters:
 *   NCC_params     : INPUT        : parameters of the MIP-NCC algorithm
 *   NCC            : INPUT        : initial NCC map of size (2*delayu+1) x (2*delayv+1), centered around the initial alignment
 *   delayu, delayv : INPUT        : vertical and horizontal half extensions of the initial NCC map 
 *   newu, newv     : INPUT        : vertical and horizontal half extensions of the output NCC map (NCC_new)
 *   ind_max        : INPUT        : linear index of maximum in NCC
 *   MIP_1, MIP_2   : INPUT        : MIPs of size dimu x dimv
 *   dimu, dimv     : INPUT        : vertical and horizontal extensions of MIP_1 and MIP_2
 *   NCCnew         : INPUT/OUTPUT : output (initially empty) NCC map of size (2*newu+1) x (2*newv+1); it extends NCC
 *   du, dv         : OUTPUT       : vertical and horizontal relative positions of the maximum of NCCnew with respect to the initial alignment
 *                                   (they are undefined if 'failed' is true)
 *   failed         : INPUT/OUTPUT : initialized to false, it is changed to true if the maximum is not centered in NCCnew
 */
void compute_Neighborhood ( NCC_parms_t *NCC_params, iom::real_t *NCC, int delayu, int delayv, int newu, int newv, int ind_max,
                                                   iom::real_t *MIP_1, iom::real_t *MIP_2, int dimu, int dimv, iom::real_t *NCCnew, int &du, int &dv, bool &failed) {

	// --- CRISTIAN MOD START --- (to optimize NCC computation)
	int ph = dimu / TILE_SIDE;
	int pw = dimv / TILE_SIDE;
	int psmatrix_size = ph*pw;

	iom::real_t *ps1 = (iom::real_t *) 0;
	iom::real_t *ps2 = (iom::real_t *) 0;

	if(psmatrix_size > 0){
		ps1 = (iom::real_t *) malloc(sizeof(iom::real_t) * ph * pw);
		ps2 = (iom::real_t *) malloc(sizeof(iom::real_t) * ph * pw);
		if(!ps1 || !ps2){
			// TODO: handle malloc failures according to policy
		}
	}

	seq_cpu_compute_partial_sums(MIP_1, MIP_2, dimu, dimv, ps1, ps2);
	// --- CRISTIAN MOD END ---	
	
	// suffixes u and v denote the vertical and the horizontal dimensions, respectively
	// suffix i denotes linear indices

	int u, v, i, d; // for variables

	// 2015-03-20. Giulio. @CHANGED newu and newv are moved as parameters
	//int newu = NCC_params->wRangeThr; // vertical half dimension of NCCnew
	//int newv = NCC_params->wRangeThr; // horizontal half dimension of NCCnew

	int ind_ref; // index of the center of NCCnew

	int initu; // vertical index of first pixel of subregion of NCC to be used to initially fill NCCnew
	int initv; // horizontal index of first pixel of subregion of NCC to be used to initially fill NCCnew
	int initi; // linear index of first pixel of subregion of NCC to be used to initially fill NCCnew

	int srcStartu; // vertical index of first pixel of the subregion of NCCnew to be reused when current maximum is moved to the center of NCCnew
	int srcStartv; // horizontal index of first pixel of the subregion of NCCnew to be reused when current maximum is moved to the center of NCCnew
	int srcStarti; // linear index of first pixel of the subregion of NCCnew to be reused when current maximum is moved to the center of NCCnew

	int dstStartu; // vertical index of first pixel of the subregion of NCCnew where the subregion to be used has to be copied
	int dstStartv; // horizontal index of first pixel of the subregion of NCCnew where the subregion to be used has to be copied
	int dstStarti; // linear index of first pixel of the subregion of NCCnew where the subregion to be used has to be copied

	int deltau; // vertical displacement of current maximum from the center of NCCnew
	int deltav; // horizontal displacement of current maximum from the center of NCCnew

	int firstv; // first horizontal index for copying elements of NCCnew to be reused
	int lastv;  // last horizontal index for copying elements of NCCnew to be reused

	int n_miss; // number of NCC to be computed to fill NCCnew
	int *missu = new int[(2*newu+1)*(2*newv+1)]; // list of vertical indices of NCC to be computed to fill NCCnew
	int *missv = new int[(2*newu+1)*(2*newv+1)]; // list of vertical indices of NCC to be computed to fill NCCnew

	// INITIALIZATION

	// fill NCCnew copying useful NCCs that have been already computed from NCC to NCCnew
	initu = MIN(MAX(0,ind_max/(2*delayv+1) - newu),2*(delayu - newu)); // initu is at least 2*(delayu - newu) to guarantee that NCCnew can be completely initialized
	initv = MIN(MAX(0,ind_max%(2*delayv+1) - newv),2*(delayv - newv)); // initv is at least 2*(delayv - newv) to guarantee that NCCnew can be completely initialized
	initi = initu * (2*delayv+1) + initv;
	if(initi < 0)
		throw iom::exception("CrossMIPs: negative index detected (initi)"); // Alessandro - 23/03/2013 - throw exception if initi is negative
	for ( u=0, i=0, d=0; u<(2*newu+1); u++, d+=2*(delayv-newv) ) // when row changes 2*(delayv-newv) values have to be skipped
		for ( v=0; v<(2*newv+1); v++ , i++)
			NCCnew[i] = NCC[i + initi + d];
	// compute displacement of the center of NCCnew with respect to the initial alignment (center of NCC)
	du = initu - delayu + newu; // displacement of first row + half dimension of NCCnew
	dv = initv - delayv + newv; // displacement of first column + half dimension of NCCnew
	// update ind_max with respect to NCCnew
	//      contribution due to rows (integer division is not distributive)               contribution due to columns
	ind_max = (2*newv+1) * (ind_max/(2*delayv+1) - initi/(2*delayv+1))   +   (ind_max%(2*delayv+1)) - (initi%(2*delayv+1));
	// index of the center of the new NCC
	ind_ref = (2*newv+1) * newu + newv;

	// UPDATE NEIGHBORHOOD AND SEARCH MAXIMUM

	int c=0; // NCC_params->maxIter iterations are allowed
	while ( c < NCC_params->maxIter && ind_max != ind_ref ) {
		// update NCCnew
		srcStartu = MAX(0,ind_max/(2*newv+1) - newu);
		srcStartv = MAX(0,ind_max%(2*newv+1) - newv);
		srcStarti = srcStartu * (2*newv+1) + srcStartv;
		deltau = ind_max/(2*newv+1) - ind_ref/(2*newv+1);
		deltav = ind_max%(2*newv+1) - ind_ref%(2*newv+1);
		dstStartu = srcStartu - deltau;
		dstStartv = srcStartv - deltav;
		dstStarti = dstStartu * (2*newv+1) + dstStartv;
		if ( srcStartu > 0 ) {     // forward copy of rows is safe
			if ( srcStartv > 0 ) { // forward copy of columns is safe
				i = 0; // first index of first row to be moved
				for ( u=0; u<((2*newu+1)-abs(deltau)); u++, i+=abs(deltav) ) { // when row changes |deltav| values have to be skipped forward
					for ( v=0; v<((2*newv+1)-abs(deltav)); v++ , i++) {
						NCCnew[i + dstStarti] = NCCnew[i + srcStarti];
					}
				}
			}
			else { // srcStartv == 0: backward copy of columns is safe
				i = (2*newv+1) - abs(deltav) - 1;  // last index of first row to be moved  
				for ( u=0; u<((2*newu+1)-abs(deltau)); u++, i+=(2*(2*newv+1) - abs(deltav)) ) { // when row changes two rows - |deltav| have to be skipped forward
					for ( v=0; v<((2*newv+1)-abs(deltav)); v++ , i--) {
						NCCnew[i + dstStarti] = NCCnew[i + srcStarti];
					}
				}
			}
		}
		else { // srcStartu == 0:     backward copy of rows is safe
			if ( srcStartv > 0 ) { // forward copy of columns is safe
				i = ((2*newu+1)-abs(deltau)-1)*(2*newv+1);     // first index last row to be moved
				for ( u=0; u<((2*newu+1)-abs(deltau)); u++, i-=(2*(2*newv+1) - abs(deltav)) ) { // when row changes two rows - |deltav| values have to be skipped backward
					for ( v=0; v<((2*newv+1)-abs(deltav)); v++ , i++) {
						NCCnew[i + dstStarti] = NCCnew[i + srcStarti];
					}
				}
			}
			else { // srcStartv == 0: backward copy of columns is safe
				i = ((2*newu+1)-abs(deltau))*(2*newv+1) - abs(deltav) - 1;     // last index of last rows to be moved
				for ( u=0; u<((2*newu+1)-abs(deltau)); u++, i-=abs(deltav) ) { // when row changes |deltav| values have to be skipped backward
					for ( v=0; v<((2*newv+1)-abs(deltav)); v++ , i--) {
						NCCnew[i + dstStarti] = NCCnew[i + srcStarti];
					}
				}
			}
		}

		// update position of the new center (current maximum)
		du += deltau;
		dv += deltav;

		// generate list of missing NCCs
		n_miss = 0;
		for ( u=0; u<(2*newu+1); u++ ) {
			if ( u+deltau < 0 || u+deltau >= (2*newu+1) ) { // all this row has to be computed
					firstv = 0;
					lastv  = 2*newv+1;
			}
			else { // only a fraction of the row has to be computed
					if ( deltav < 0 ) { // the initial part of the row has to be computed
							firstv = 0;
							lastv  = -deltav;
					}
					else if ( deltav > 0 ) { // the final part of the row has to be computed
							firstv = (2*newv+1) - deltav;
							lastv  = 2*newv+1;
					}
					else { // deltav == 0: no NCCs has to be computed
							firstv = 0;
							lastv  = 0;
					}
			}
			for ( v=firstv; v<lastv; v++ ) {
					missu[n_miss] = u;
					missv[n_miss] = v;
					n_miss++;
			}
		}
		// CHECK, MUST BE: n_miss == ((2*newu+1)*(2*newv+1) - ((2*newu+1)-abs(deltau))*((2*newv+1)-abs(deltav)))
		if ( n_miss != ((2*newu+1)*(2*newv+1) - ((2*newu+1)-abs(deltau))*((2*newv+1)-abs(deltav))) )
			throw iom::exception("CrossMIPs: incomplete NCC map in compute_Neighborhood");

		// compute missing NCCs
#if defined(USECUDA)
		if(usecuda>0) { // use the CUDA card
			unsigned int nthreads=NTHREADS;
			unsigned int nblocks=n_miss;
			// computes means
			if((2*newu+1)*(2*newv+1) > sizerv){
				// 2018-06-05. Giulio. @FIXED reallocation and initialization of d_missrv moved from here because dimu and/or dimv could have been changed
				if(sizerv > 0){
					MY_CUDA_CHECK(hipFree(d_missrv));
				}
				sizerv = (2*newu+1)*(2*newv+1);
				MY_CUDA_CHECK (
					hipMalloc (( void **) &d_missrv, sizerv*sizeof ( iom::real_t ) ));
				MY_CUDA_CHECK ( hipMemset (d_missrv, 0, sizeof(iom::real_t)*sizerv));
			}
			if(sizemiss<n_miss) {
			   if(sizemiss>0) {
				   MY_CUDA_CHECK(hipFree(d_missu));
				   MY_CUDA_CHECK(hipFree(d_missv));
			   }
			   sizemiss=n_miss;
		   
			   MY_CUDA_CHECK (hipMalloc (( void **) &d_missu, sizemiss*sizeof(int)));
			   MY_CUDA_CHECK (hipMalloc (( void **) &d_missv, sizemiss*sizeof(int)));
			}
		
			MY_CUDA_CHECK( hipMemcpy(d_missu, missu,
						  sizeof(int)*n_miss,hipMemcpyHostToDevice) );
			MY_CUDA_CHECK( hipMemcpy(d_missv, missv,
						  sizeof(int)*n_miss,hipMemcpyHostToDevice) );
					  
			if(sizeimg<(dimu*dimv)) {
				fprintf(stderr,"Reallocating memory before gpu_NCC_miss: %d, now is %d\n",
				sizeimg,(dimu*dimv));
				assert(sizeimg);
				MY_CUDA_CHECK(hipFree(d_im1));
				MY_CUDA_CHECK(hipFree(d_im2));
				sizeimg=(dimu)*(dimv);
				MY_CUDA_CHECK (
					hipMalloc (( void **) &d_im1, sizeimg*sizeof ( iom::real_t ) ));
				MY_CUDA_CHECK (
					hipMalloc (( void **) &d_im2, sizeimg*sizeof ( iom::real_t ) ));
			}
			if(s_im1!=MIP_1) {
				MY_CUDA_CHECK( hipMemcpy(d_im1, MIP_1,
						   sizeof(iom::real_t)*(dimu*dimv), hipMemcpyHostToDevice) );
				s_im1=MIP_1;
			}
			if(s_im2!=MIP_2) {
				MY_CUDA_CHECK( hipMemcpy(d_im2, MIP_2,
						   sizeof(iom::real_t)*(dimu*dimv), hipMemcpyHostToDevice) );
				s_im2=MIP_2;
			}
			//fprintf(stderr,"----> %p %lu %d %d\n",d_missrv,sizeof(iom::real_t),newu,newv); // IANNELLO
			MY_CUDA_CHECK( hipMemcpy(d_missrv, NCCnew,
				   sizeof(iom::real_t)*(2*newu+1)*(2*newv+1), hipMemcpyHostToDevice) );
			gpu_NCC_miss<<<nblocks, nthreads>>>
							(d_im1, d_im2, d_missrv, dimu, dimv,
							 du, dv, newu, newv, d_missu, d_missv);
			MY_CUDA_CHECK( hipMemcpy(NCCnew, d_missrv,
				   sizeof(iom::real_t)*(2*newu+1)*(2*newv+1), hipMemcpyDeviceToHost) );
		} 
		else 
#endif // defined(USECUDA)
		{ // 2019-04-25. Giulio. This code should always be executed if CUDA code is not enabled
			//fprintf(stderr,"----> %lu %d %d\n",sizeof(iom::real_t),newu,newv); // IANNELLO
			for ( i=0; i<n_miss; i++ ) {
				// indices over MIPs have to be shifted to take into account their relative position with respecto to the center of NCCnew
				// and the relative position of the center with respect to the initial initial alignment (center of NCC)
				u = missu[i] - newu + du;
				v = missv[i] - newv + dv;
				NCCnew[missu[i]*(2*newv+1)+missv[i]] = compute_NCC(MIP_1, MIP_2,dimu,dimv, u, v, ps1, ps2);
			}
		}
#if defined(DUMPNCC)
		for ( i=0; i<n_miss; i++ ) {
			fprintf(stderr,"NCCnew[%d]=%f\n",missu[i]*(2*newv+1)+missv[i],
										NCCnew[missu[i]*(2*newv+1)+missv[i]]);
		}
#endif
		// find maximum
		ind_max = compute_MAX_ind(NCCnew,(2*newu+1)*(2*newv+1));

		c++;
	}

	if(ind_ref != ind_max)
	{
		deltau = ind_max/(2*newv+1) - ind_ref/(2*newv+1);
		deltav = ind_max%(2*newv+1) - ind_ref%(2*newv+1);

		// update position of the new center (current maximum)
		du += deltau;
		dv += deltav;

		failed=true;
	}

	// --- CRISTIAN MOD START ---
	if(ps1) 
		delete ps1;
	if(ps2) 
		delete ps2;
	// --- CRISTIAN MOD END ---

	delete[] missu;
	delete[] missv;
}


//void compute_Alignment( NCC_parms_t *NCC_params, REAL_T *NCC_xy, REAL_T *NCC_xz, REAL_T *NCC_yz,
//                                          int dimi, int dimj, int dimk, int ind_xy, int ind_xz, int ind_yz, NCC_descr_t *result) {
void compute_Alignment( NCC_parms_t *NCC_params, iom::real_t *NCC_xy, iom::real_t *NCC_xz, iom::real_t *NCC_yz,
                                            int dimi, int dimj, int dimk, int dx1, int dx2, int dy1, int dy2, int dz1, int dz2, bool failed_xy, bool failed_xz, bool failed_yz, NCC_descr_t *result) {

	int w1x, w2x, w1y, w2y, w1z, w2z;

	compute_NCC_width(NCC_params,NCC_xy,(2*dimj+1),(dimi*(2*dimj+1)+dimj),NCC_params->wRangeThr_i,NCC_params->wRangeThr_j,failed_xy, w1x,w1y);
	compute_NCC_width(NCC_params,NCC_xz,(2*dimk+1),(dimi*(2*dimk+1)+dimk),NCC_params->wRangeThr_i,NCC_params->wRangeThr_k,failed_xz, w2x,w1z);
	compute_NCC_width(NCC_params,NCC_yz,(2*dimk+1),(dimj*(2*dimk+1)+dimk),NCC_params->wRangeThr_j,NCC_params->wRangeThr_k,failed_yz, w2y,w2z);

	compute_NCC_alignment(NCC_params,result,0,dx1,NCC_xy[(dimi*(2*dimj+1)+dimj)],w1x,dx2,NCC_xz[(dimi*(2*dimk+1)+dimk)],w2x);
	compute_NCC_alignment(NCC_params,result,1,dy1,NCC_xy[(dimi*(2*dimj+1)+dimj)],w1y,dy2,NCC_yz[(dimj*(2*dimk+1)+dimk)],w2y);
	compute_NCC_alignment(NCC_params,result,2,dz1,NCC_xz[(dimi*(2*dimk+1)+dimk)],w1z,dz2,NCC_yz[(dimj*(2*dimk+1)+dimk)],w2z);
}


void enhance ( iom::real_t *im, int imLen, int graylevels, NCC_parms_t *NCC_params ) {
/*
 * the enhancement transformation is a multi-linear curve with n_transforms rescaled linear
 * tranformations
 * for i=0, ..., (n_transforms-1), percentiles[i] contains the fraction of pixels to which the
 * (i+1)-th rescaled linear transformation has to be applied; percentiles[n_transforms-1] must be 1.00
 * for i=1, ..., n_transforms, c[i-1] amd c[i] contain the lowest and highest value corresponding to
 * the the i-th rescaled linear transformation; c[0] must be 0.00 and c[n_transforms] must be 1.00
 */

	int n_transforms = NCC_params->n_transforms;
	iom::real_t *percentiles = NCC_params->percents;
	iom::real_t *c = NCC_params->c; // tranformed values of thresholds
	iom::real_t *thresholds = new iom::real_t[n_transforms+1];
	iom::real_t *a = new iom::real_t[n_transforms+1];
	iom::real_t *b = new iom::real_t[n_transforms+1];
	int i, j;
	bool found;

	thresholds[0] = (iom::real_t)0.00;

	stack_percentiles(im,imLen,graylevels,percentiles,thresholds,n_transforms);

	for ( i=1; i<=n_transforms; i++ ) {
			a[i] = (c[i] - c[i-1]) / (thresholds[i] - thresholds[i-1]);
			b[i] = c[i] - a[i]*thresholds[i];
	}

	for ( i=0; i<imLen; i++ ) {
			binary_search(thresholds,n_transforms,im[i],found,j);
			im[i] = a[j]*im[i] + b[j];
	}

	delete[] thresholds;
	delete[] a;
	delete[] b;
}


void stack_percentiles ( iom::real_t *im, int imLen, int graylevels,
                                                 iom::real_t *percentiles, iom::real_t *thresholds, int n_percentiles ) {

	iom::real_t d = (iom::real_t)1.0 / (iom::real_t)graylevels;
	iom::real_t *cumsum = new iom::real_t[graylevels];
	int i, j;

	for ( i=0; i<graylevels; i++ )
		cumsum[i] = 0;

	for ( i=0; i<imLen; i++ ) {
		j = MIN((int)floor(im[i]/d),graylevels-1); // guarantees that index is within limits
		cumsum[j]++;
	}

	cumsum[0] /= imLen;
	for ( i=1; i<graylevels; i++ )
		cumsum[i] = cumsum[i]/imLen + cumsum[i-1];

	// guarantees that last cumulative fraction is 1.0
	cumsum[graylevels-1] = 1.0;

	for ( i=0, j=1; j<n_percentiles; ) {
		if ( i == graylevels ) {
			DISPLAY_ERROR("i out of limits");
			exit(1);
		}
		if ( cumsum[i] >= percentiles[j-1] ) {
			thresholds[j] = d * i;
			j++;
		}
		else
			i++;
	}
	thresholds[n_percentiles] = (iom::real_t)1; // to avoid round off errors

	delete[] cumsum;
}
